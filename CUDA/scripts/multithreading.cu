#include <hip/hip_runtime.h>
#include <iostream>

#define CHECK_CUDA(call)                                                   \
  {                                                                        \
    hipError_t err = call;                                                \
    if (err != hipSuccess) {                                              \
      std::cerr << "CUDA Error: " << hipGetErrorString(err)               \
                << " at " << __FILE__ << ":" << __LINE__ << std::endl;     \
      std::exit(EXIT_FAILURE);                                             \
    }                                                                      \
  }

// CUDA kernel: each thread adds one element
__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    const int N = 1 << 20;           // 1M elements
    const size_t bytes = N * sizeof(float);

    // 1) Allocate host memory
    float *h_A = (float*)malloc(bytes);
    float *h_B = (float*)malloc(bytes);
    float *h_C = (float*)malloc(bytes);

    // 2) Initialize inputs
    for (int i = 0; i < N; ++i) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(2*i);
    }

    // 3) Allocate device memory
    float *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc(&d_A, bytes));
    CHECK_CUDA(hipMalloc(&d_B, bytes));
    CHECK_CUDA(hipMalloc(&d_C, bytes));

    // 4) Copy data to device
    CHECK_CUDA(hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice));

    // 5) Launch kernel with enough threads to cover N elements
    int threadsPerBlock = 256;
    int blocksPerGrid   = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());

    // 6) Copy result back to host
    CHECK_CUDA(hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost));

    // 7) Verify a few results
    bool ok = true;
    for (int i = 0; i < 5; ++i) {
        float expected = h_A[i] + h_B[i];
        if (fabs(h_C[i] - expected) > 1e-5) {
            ok = false;
            break;
        }
    }
    std::cout << (ok ? "PASS\n" : "FAIL\n");

    // 8) Clean up
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);

    return 0;
}
