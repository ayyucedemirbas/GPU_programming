#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel for vector addition
__global__ void vec_add(float *a, float *b, float *c, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    int n = 10;
    float a[n], b[n], c[n];
    float *d_a, *d_b, *d_c;

    // Initialize input vectors
    for (int i = 0; i < n; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    // Allocate GPU memory
    hipMalloc(&d_a, n * sizeof(float));
    hipMalloc(&d_b, n * sizeof(float));
    hipMalloc(&d_c, n * sizeof(float));

    // Copy data to GPU
    hipMemcpy(d_a, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel (1 block, 256 threads per block)
    int block_size = 256;
    int grid_size = (n + block_size - 1) / block_size;
    vec_add<<<grid_size, block_size>>>(d_a, d_b, d_c, n);

    // Copy result back to CPU
    hipMemcpy(c, d_c, n * sizeof(float), hipMemcpyDeviceToHost);

    // Print results
    printf("A: ");
    for (int i = 0; i < n; i++) printf("%.2f ", a[i]);
    printf("\nB: ");
    for (int i = 0; i < n; i++) printf("%.2f ", b[i]);
    printf("\nC: ");
    for (int i = 0; i < n; i++) printf("%.2f ", c[i]);

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}